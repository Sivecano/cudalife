#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "SDL2/SDL.h"
#include "hiprand.h"
#include <SDL2/SDL_keycode.h>
#include <SDL2/SDL_render.h>

//#define pos(x, y) (x + 1920*y)

const dim3 threads(128, 8, 1);
const dim3 blocks(15, 135, 1);

#define n_colour 25

__device__ uint32_t pos(uint32_t x, uint32_t y)
{
    return (x + 1920*y);
}

__global__ void solidify(uint8_t* arr)
{
    unsigned int ind = pos(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    arr[ind] %= n_colour;
    return;
    if (arr[ind] % 2 == 1)
        arr[ind] = 0xff;
    else
        arr[ind] = 0;
}

__global__ void clear(uint8_t* arr)
{
  arr[pos(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y)] = 0;
}

__global__ void conway(const uint8_t* in, uint8_t* out)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    int neighbors = 0;

    for (int i = - (x > 0 ? 1 : 0); i < 1 + (x < 1919 ? 1 : 0); i++)
        for (int j = - (y > 0 ? 1 : 0); j < 1 + (y < 1079 ? 1 : 0); j++)
            if (in[pos(x + i, y + j)] % 2)
                neighbors++;
    

    if (in[pos(x, y)] > 0)
        neighbors--;


    if (neighbors == 2)
        out[pos(x, y)] = in[pos(x, y)];

    else if (neighbors == 3)
        out[pos(x, y)] = 1;
    else out[pos(x,y)] = 0;

    //if (out[pos(x,y)]) out[pos(x-2,y+2)] = 0xff;
}

__global__ void cyclic(const uint8_t* in, uint8_t* out)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    uint8_t cur = in[pos(x, y)];
    uint8_t target = (cur + 1) % n_colour;
    out[pos(x, y)] = cur;

    for (int i = - (x > 0 ? 1 : 0); i < 1 + (x < 1919 ? 1 : 0); i++)
        for (int j = - (y > 0 ? 1 : 0); j < 1 + (y < 1079 ? 1 : 0); j++)
            if (in[pos(x + i, y + j)] == target)
                out[pos(x, y)] = target;
}

__global__ void cyclic_bugged(const uint8_t* in, uint8_t* out)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    uint8_t cur = in[pos(x, y)];
    uint8_t target = (cur + 1) % n_colour;
    out[pos(x, y)] = cur;

    for (int i = - (x > 0 ? 1 : 0); i < 1 + (x < 1919 ? 1 : 0); i++)
        for (int j = - (y > 0 ? 1 : 0); j < 1 + (y < 1079 ? 1 : 0); j++)
            if (i != 0 && j != 0 && in[pos(x + i, y + j)] == target)
                out[pos(x, y)] = target;
}

__global__ void draw(const uint8_t* in, uint32_t* out)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
    if(in[pos(x, y)])
        out[pos(x,y)] = 0x00006400;
    else
        out[pos(x, y)] = 0xff1a1a1a;
}

__global__ void cdraw(const uint8_t* in, uint32_t* out)
{
    const uint32_t colormap[] = {0xffbf3f3f, 0xffbf663f, 0xffbf8c3f, 0xffbfb23f, 0xffa5bf3f, 0xff7fbf3f, 0xff59bf3f, 0xff3fbf4c, 0xff3fbf72, 0xff3fbf99, 0xff3fbfbf, 0xff3f99bf, 0xff3f72bf, 0xff3f4cbf, 0xff593fbf, 0xff7f3fbf, 0xffa53fbf, 0xffbf3fb2, 0xffbf3f8c, 0xffbf3f66, 0xffbf3f3f, 0xffbf663f, 0xffbf8c3f, 0xffbfb23f, 0xffa5bf3f, 0xff7fbf3f, 0xff59bf3f, 0xff3fbf4c, 0xff3fbf72, 0xff3fbf99, 0xff3fbfbf, 0xff3f99bf, 0xff3f72bf, 0xff3f4cbf, 0xff3f3fbf, 0xff3f3fbf, 0xff3f3fbf, 0xff3f3fb2, 0xff3f3f8c, 0xff3f3f66};

    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
    out[pos(x,y)] = colormap[in[pos(x,y)] % n_colour];
}



int main(int argc, char* argv[])
{
    SDL_Init(SDL_INIT_EVERYTHING);

    uint8_t* board;
    uint8_t* buffer;
    uint32_t* colours;
    hiprandGenerator_t gen;
    if (hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW) !=HIPRAND_STATUS_SUCCESS) SDL_Log("WE FUCKED UP: %s", hipGetErrorString(hipGetLastError()));
    hiprandSetPseudoRandomGeneratorSeed(gen, rand());
    hipError_t err;
    
    SDL_Log("set cuda device");
    if (hipSetDevice(0) != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }
    
    SDL_Log("Window init");

    SDL_Window* win = SDL_CreateWindow("conway's game of cuda", 0, 0, 1920, 1080,
                                       SDL_WINDOW_INPUT_FOCUS | SDL_WINDOW_MOUSE_FOCUS |
                                       0 | SDL_WINDOW_ALLOW_HIGHDPI |
                                       SDL_WINDOW_SKIP_TASKBAR | SDL_WINDOW_SHOWN );
    SDL_Renderer* ren = SDL_CreateRenderer(win, 0, 0);

    SDL_Log("we rendering");
    SDL_Surface* sur = SDL_GetWindowSurface(win);
    SDL_Log("surdim : %i, surf: bpp: %i, fmt: %i, %s, h: %i, pitch: %i, pitch/4 : %f", sur->pitch * sur->h / 4, sur->format->BitsPerPixel, sur->format->format, SDL_GetPixelFormatName(sur->format->format), sur->h, sur->pitch, sur->pitch / 4.);

    err = hipMalloc(&board, 1080 * 1920); if (err != hipSuccess) SDL_Log("%s", hipGetErrorString(err));
    err = hipMalloc(&buffer, 1080 * 1920); if (err != hipSuccess) SDL_Log("%s", hipGetErrorString(err));
    err = hipMalloc(&colours, 4 * 1080 * 1920); if (err != hipSuccess) SDL_Log("%s", hipGetErrorString(err));

    if (hiprandGenerate(gen, reinterpret_cast<unsigned int *>(board), 1080 * 1920 / 4) !=HIPRAND_STATUS_SUCCESS) SDL_Log("RANDOM FUCKED UP");
    solidify<<<blocks, threads>>>(board);
    err = hipDeviceSynchronize();
    if (err != hipSuccess) SDL_Log("%s", hipGetErrorString(err));



    bool running = true;
    bool pause = true;
    // int time = 0;
    uint8_t delay = 0;

    while (running)
    {
        // time = SDL_GetTicks();
        SDL_Event e;
        while(SDL_PollEvent(&e)) {
            switch(e.type) {
                case SDL_QUIT:
                    running = false;
                    break;
                case SDL_KEYUP:
                    switch (e.key.keysym.sym) {
                        case SDLK_r:
                            SDL_Log("rand");
                            if (hiprandGenerate(gen, reinterpret_cast<unsigned int *>(board), 1080 * 1920 / 4) !=
                                HIPRAND_STATUS_SUCCESS)
                                SDL_Log("RANDOM FUCKED UP");
                            solidify<<<blocks, threads>>>(board);
                            err = hipDeviceSynchronize();
                            if (err != hipSuccess) SDL_Log("%s", hipGetErrorString(err));
                            break;

                        case SDLK_SPACE:
                            pause = !pause;
                            break;

                        case SDLK_q:
                            running = false;
                            break;

                        case SDLK_c:
                            hipMemset(board, 0, 1920 * 1080);
                            break;
                        
                        case SDLK_UP:
                            delay += 1;
                            break;
                            
                        case SDLK_DOWN:
                            delay -= 1;
                            break;

                        default:
                            break;

                    }
                    break;

                case SDL_MOUSEMOTION:
                    if (e.motion.state & SDL_BUTTON_RMASK) {
                        err = hipMemset(board + e.motion.x + 1920 * e.motion.y, 0x00, 2);
                        if (err != hipSuccess) SDL_Log("%s", hipGetErrorString(err));
                    } else if (e.motion.state & SDL_BUTTON_LMASK) {
                        err = hipMemset(board + e.motion.x + 1920 * e.motion.y, 0xff, 2);
                        if (err != hipSuccess) SDL_Log("%s", hipGetErrorString(err));
                    }
                    break;

                default:
                    break;
            }
        }
        //SDL_Log("before");
        if (!pause) {
            clear<<<blocks, threads>>>(buffer);
            cyclic_bugged<<<blocks, threads>>>(board, buffer);
            err = hipDeviceSynchronize();
            if (err != hipSuccess) SDL_Log("%s", hipGetErrorString(err));
            //SDL_Log("%s", hipGetErrorString(hipGetLastError()));
            uint8_t* temp = buffer;
            buffer = board;
            board = temp;
        }
        
        
        cdraw<<<blocks, threads>>>(board, colours);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) 
            SDL_Log("%s", hipGetErrorString(err));
        //SDL_Log("helo");
        SDL_LockSurface(sur);
        if (hipMemcpy(sur->pixels, colours, 4* 1080 * 1920, hipMemcpyDeviceToHost) != hipSuccess)
            SDL_Log("Oops!");
        SDL_UpdateWindowSurface(win);
        SDL_UnlockSurface(sur);
        // SDL_SetRenderDrawColor(ren, 255, 0, 0, 0);
        // SDL_RenderClear(ren);
        // SDL_RenderPresent(ren);
        // while (SDL_GetTicks() - time < 4);
        SDL_Delay(delay);
        // SDL_Log("here\n\n");
        // SDL_Log("frametime: %i", SDL_GetTicks() - time);
    }

    hiprandDestroyGenerator(gen);
    hipFree(board);
    hipFree(buffer);
    hipFree(colours);
    SDL_DestroyWindow(win);
    SDL_Quit();

    return 0;
}
